#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <time.h>  //Для clock_gettime
#include <fstream>
#include <iostream>
#include <complex>
#include <stdlib.h>
#include <unistd.h>
//#include <omp.h>

#define REDUCTIONTYPE 1//0-mutex 1-параллельно
#define DIFRACTION 1
#define CompCount 6 //комплексный вектор xyz  3-Re, 3-Im
#define MAXCONSTPARAM 5

#define FLOATTYPE 0//0-FLOAT, 1-float

#define NBLOCKS 80
#define NTHREADS 96



using namespace std;
#include "LibIntGPU.cu"



int main()
{
	SetDevice(0);

	struct timespec mt1, mt2; 
	long int tt;      
	ofstream FileRes;
	ofstream FileTempRes;
	
	struct stat st={0};
	if(stat("./Results", &st)==-1)
		mkdir("./Results", 0700);
	
	FileRes.open("./Results/ResPower",std::ios::out);
	
	int N_theta=300;
	int N_phi=100;
	
	FLOAT d_theta=M_PI/FLOAT(N_theta);
	FLOAT d_phi=2.*M_PI/FLOAT(N_phi);
	
	FLOAT Alpha=0, theta=0, phi=0;
	
	FLOAT Sigma0_1=1.86,Sigma0_2=5.28;
	FLOAT w0=25.46;
	FLOAT E1=0.1775;
	FLOAT E2=0.2531;
	FLOAT k=sqrt(3);
	FLOAT eps=3./4.;
	FLOAT R1=w0*Sigma0_1*Sigma0_1*0.5, R2=w0*Sigma0_2*Sigma0_2*0.5;//длины релея
	
	FLOAT R=4.*Sigma0_1;
	FLOAT z0=150.;

	FLOAT IntParams[9]={-R, R, 100, -R, R, 100, -z0, z0, 1500};
	
	int NumParam=8; //число переменных параметров
    FLOAT FuncParam[8]={Alpha, theta, phi, sin(Alpha), cos(Alpha), sin(Alpha)*sin(Alpha), cos(Alpha)*cos(Alpha), cos(2.*Alpha)};
    
   // int NumCoParam=MAXCONSTPARAM; //число постоянных параметров
    FLOAT co_FuncParam[5]={R1, R2, Sigma0_1*Sigma0_1, Sigma0_2*Sigma0_2, k};
    CopyToConstantMem(MAXCONSTPARAM, co_FuncParam);
    
    int count=CompCount;
	FLOAT *J_res;
	J_res=new FLOAT[count];
    
	FLOAT *B;
	B=new FLOAT[count];
	
	for (int i=0; i<count; i++)
	{	J_res[i]=0;
		B[i]=0;
	}
    
	FLOAT Int=0.;
	FLOAT tempPower=0.;
	FLOAT Power=0;
	FLOAT PowerCoeff=0.69/(8.*sqrt(eps)*M_PI*M_PI);
	FLOAT Coeff=E1*E2*k;
	
	ostringstream strs;
	string str;
	str="";
	FileRes<<"Alpha	Power	time(sec)"<<endl;
	for(Alpha=0; Alpha<M_PI_2; Alpha+=0.087266463) //шаг 5 градусов
	{
		printf("Alpha %f \n", Alpha);
		
		clock_gettime (CLOCK_REALTIME, &mt1);
		
		FuncParam[0]=Alpha;
		FuncParam[3]=sin(Alpha);
		FuncParam[4]=cos(Alpha);
		FuncParam[5]=sin(Alpha)*sin(Alpha);
		FuncParam[6]=cos(Alpha)*cos(Alpha);
		FuncParam[7]=cos(2.*Alpha);
		
		strs<<Alpha/M_PI*180.;
		str = strs.str();
		FileTempRes.open(("./Results/ResPowerAlpha0"+str).c_str(),std::ios::out);
		strs.str("");
		strs.clear();
		
		FileTempRes<<"i_theta"<<"	"<<"theta"<<"	"<<"i_phi"<<"	"<<"phi"<<"	"<<"Bx.real()"<<"	"<<"Bx.imag()"<<"	"<<"By.real()"<<"	"<<"By.imag()"<<" "<<"Bz.real()"<<"	"<<"Bz.imag()"<<"	"<<"Int"<<"	"<<"Int*PowerCoeff"<<endl;
	
		for(int i_theta=0;i_theta<N_theta;i_theta++)
		{
			theta=i_theta*d_theta+0.5*d_theta;
			FuncParam[1]=theta;
			
			for(int i_phi=0; i_phi<N_phi; i_phi++) 
			{	
				phi=i_phi*d_phi+0.5*d_phi-M_PI_2;
				FuncParam[2]=phi;
				
				ParallelNquadIntegrator(3,IntParams, NumParam, FuncParam, J_res);
				printf("i %d, j %d, theta %f, phi %f, Re(J_res.x) %f, Im(J_res.x) %f, Re(J_res.y) %f, Im(J_res.y) %f, Re(J_res.z) %f, Im(J_res.z) %f \n", i_theta, i_phi, theta, phi, J_res[0], J_res[0+3], J_res[1], J_res[1+3], J_res[2], J_res[2+3]);
				
				B[0]=(J_res[2]*sin(phi)*sin(theta)-J_res[1]*cos(theta))*Coeff; //Bx real
				B[1]=-(J_res[0]*cos(theta)-J_res[2]*cos(phi)*sin(theta))*Coeff; //By real
				B[2]=(J_res[1]*cos(phi)*sin(theta)-J_res[0]*sin(phi)*sin(theta))*Coeff; //Bz real
				
				B[0+3]=(J_res[2+3]*sin(phi)*sin(theta)-J_res[1+3]*cos(theta))*Coeff; //Bx imag
				B[1+3]=-(J_res[0+3]*cos(theta)-J_res[2+3]*cos(phi)*sin(theta))*Coeff; //By imag
				B[2+3]=(J_res[1+3]*cos(phi)*sin(theta)-J_res[0+3]*sin(phi)*sin(theta))*Coeff; //Bz imag
				
				
				Int=B[0]*B[0]+B[1]*B[1]+B[2]*B[2]+B[0+3]*B[0+3]+B[1+3]*B[1+3]+B[2+3]*B[2+3];
				tempPower=Int*PowerCoeff*sin(theta)*d_theta*d_phi;
				Power+=tempPower;
			
				FileTempRes<<i_theta<<"	"<<theta<<"	"<<i_phi<<"	"<<phi<<"	"<<B[0]<<"	"<<B[0+3]<<"	"<<B[1]<<" "<<B[1+3]<<"	"<<B[2]<<"	"<<B[2+3]<<"	"<<Int<<"	"<<tempPower<<"	"<<Power<<endl;
			}
			
		}
		
		cout<<"angle="<<Alpha/M_PI*180.<<"	"<<"Power="<<Power<<endl;

		
		clock_gettime (CLOCK_REALTIME, &mt2);

		//Рассчитываем разницу времени между двумя измерениями
		tt=1000000000*(mt2.tv_sec - mt1.tv_sec)+(mt2.tv_nsec - mt1.tv_nsec);
		FileRes<<Alpha/M_PI*180.<<"	"<<Power<<"	"<<tt/1000000000.<<endl;
		Power=0;

		//Выводим результат расчета на экран
		printf ("time: %g с\n",tt/1000000000.);
		
		FileTempRes.close();
	}
	


	FileRes.close();


	delete J_res;
	delete B;
	
	return 0;
	

}

