#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
//#include <python3.6/Python.h>



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#if FLOATTYPE==0
	#define FLOAT double
	#define cuFLOATComplex  hipDoubleComplex
	#define make_cuFLOATComplex  make_hipDoubleComplex
#endif

#if FLOATTYPE==1
	#define FLOAT float
	#define cuFLOATComplex  hipFloatComplex
	#define make_cuFLOATComplex  make_hipFloatComplex
	#define hipCmul hipCmulf
	#define hipCadd hipCaddf
	#define hipCdiv hipCdivf
	#define hipCreal hipCrealf
	#define hipCimag hipCimagf

#endif


struct cuComplex3
{
	cuFLOATComplex x, y, z;
};



int getSPcores(hipDeviceProp_t devProp)
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
     case 2: // Fermi
      if (devProp.minor == 1) cores = mp * 48;
      else cores = mp * 32;
      break;
     case 3: // Kepler
      cores = mp * 192;
      break;
     case 5: // Maxwell
      cores = mp * 128;
      break;
     case 6: // Pascal
      if (devProp.minor == 1) cores = mp * 128;
      else if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 7: // Volta
      if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     default:
      printf("Unknown device type\n"); 
      break;
      }
    return cores;
}

int assigned_device;
void SetDevice(int gpu)
{
  assigned_device=gpu;

int used_device;
  // Select the used device:
  if ( hipSetDevice(assigned_device) != hipSuccess or
      hipGetDevice( &used_device ) != hipSuccess or
      used_device != assigned_device) {
    printf ("Error: unable to set device %d\n", assigned_device);
  }
hipDeviceProp_t prop;

hipGetDeviceProperties(&prop,assigned_device);
  int cores;
  cores=getSPcores(prop);
  cout<<"multiProcessorCount = "<<prop.multiProcessorCount<<endl;
  cout<<"maxThreadsPerBlock = "<< prop.maxThreadsPerBlock<<endl;
  cout<<"maxGridSize[0] = "<< prop.maxGridSize[0]<<endl;
  cout<<"l2CacheSize = "<<prop.l2CacheSize<<endl;
  cout<<"regsPerBlock = "<<prop.regsPerBlock<<endl;
  cout<<"memoryClockRate = "<<prop.memoryClockRate<<endl;
  cout<<"sharedMemPerBlock = "<<prop.sharedMemPerBlock<<endl;
  cout<<"totalGlobalMem = "<<prop.totalGlobalMem<<endl;
  cout<<"totalConstMem = "<<prop.totalConstMem<<endl;
  cout<<"clockRate = "<<prop.clockRate<<endl;
  cout<<"name = "<<prop.name<<endl;
  cout<<"cores = "<<cores<<endl;
  cout<<endl;
  
}

__device__ __host__ cuFLOATComplex  operator*(cuFLOATComplex a, cuFLOATComplex b) { return hipCmul(a,b); }
__device__ __host__ cuFLOATComplex  operator+(cuFLOATComplex a, cuFLOATComplex b) { return hipCadd(a,b); }
__device__ __host__ cuFLOATComplex  operator/(cuFLOATComplex a, cuFLOATComplex b) { return hipCdiv(a,b); }
__device__ __host__ cuFLOATComplex  operator-(cuFLOATComplex a, cuFLOATComplex b) { return make_cuFLOATComplex(hipCreal(a)-hipCreal(b),hipCimag(a)-hipCimag(b));}
__device__ __host__ cuFLOATComplex  operator-(cuFLOATComplex a) { return make_cuFLOATComplex(-hipCreal(a),-hipCimag(a));}
//операции с обычными вещественными числами
__device__ __host__ cuFLOATComplex  operator*(cuFLOATComplex a, FLOAT b) { return make_cuFLOATComplex(hipCreal(a)*b,hipCimag(a)*b); }
__device__ __host__ cuFLOATComplex  operator/(cuFLOATComplex a, FLOAT b) { return make_cuFLOATComplex(hipCreal(a)/b,hipCimag(a)/b); }
__device__ __host__ cuFLOATComplex  operator+(cuFLOATComplex a, FLOAT b) { return make_cuFLOATComplex(hipCreal(a)+b,hipCimag(a)); }
__device__ __host__ cuFLOATComplex  operator-(cuFLOATComplex a, FLOAT b) { return make_cuFLOATComplex(hipCreal(a)-b,hipCimag(a)); }


__device__ __host__ cuFLOATComplex  operator*(FLOAT a, cuFLOATComplex b) { return make_cuFLOATComplex(hipCreal(b)*a,hipCimag(b)*a); }
__device__ __host__ cuFLOATComplex  operator/(FLOAT a, cuFLOATComplex b) { return hipCdiv(make_cuFLOATComplex(a,0),b); }
__device__ __host__ cuFLOATComplex  operator+(FLOAT a, cuFLOATComplex b) { return make_cuFLOATComplex(hipCreal(b)+a,hipCimag(b)); }
__device__ __host__ cuFLOATComplex  operator-(FLOAT a, cuFLOATComplex b) { return make_cuFLOATComplex(a-hipCreal(b),-hipCimag(b)); }

__device__ __host__ cuFLOATComplex  exp(cuFLOATComplex a) {

FLOAT x=hipCreal(a);
FLOAT y=hipCimag(a);
// exp(x) * (cos(y) + i sin(y))
	return exp(x)*(cos(y) +  make_cuFLOATComplex(0,1.)*sin(y));

	}
	

	
__constant__ FLOAT co_FuncParam[MAXCONSTPARAM];

#include "cuFuncs.cu"





#if (!defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600)
#else
#if FLOATTYPE == 0
__device__ double atomicAdd(double* address, double val) //не передавать в val конструкции типа "threadIdx.x*...", будет бред...
{
	unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +
				__longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}
#endif
#endif

extern __shared__ FLOAT shared[];



extern "C" void CopyToConstantMem(int ParamNum,FLOAT *FuncConstParam)
{
	hipMemcpyToSymbol(HIP_SYMBOL(co_FuncParam),FuncConstParam,ParamNum*sizeof(FLOAT));

}


__global__ void KernelParallelIntegrator(FLOAT *result, int multiplicity, FLOAT *IntParams, FLOAT *FuncParam, FLOAT *dX, int *TotalN, FLOAT *LeftLim) //ядро
{
	
	int indeX=blockIdx.x*blockDim.x+threadIdx.x;
	__shared__ FLOAT shRes[6];
	
	if(threadIdx.x<6)
		shRes[threadIdx.x]=0;
		__syncthreads();
	
	int count=CompCount;
	FLOAT *locresult;
	locresult=new FLOAT[count];
	for (int i=0; i<count; i++)
	locresult[i]=0;
	
	FLOAT *X;//координата по каждому интегралу
	X=new FLOAT[multiplicity];
	int *ind;
	ind=new int[multiplicity];
	
	cuComplex3 tempRes;
	tempRes.x=make_cuFLOATComplex(0, 0);
	tempRes.y=make_cuFLOATComplex(0, 0);
	tempRes.z=make_cuFLOATComplex(0, 0);
	
	for(int index=indeX;index<TotalN[0];index+=blockDim.x * gridDim.x)
	{
		ind[0]=index;
		X[0]=ind[0]/TotalN[1];
		for(int n=1;n<multiplicity;n++)//для каждого индекса вычислить соответствующий отрезок интегрирования
		{
			ind[n]=ind[n-1]-X[n-1]*TotalN[n];
			X[n]=ind[n]/TotalN[n+1];
		}
		for(int n=0;n<multiplicity;n++)
		{
			X[n]=LeftLim[n]+X[n]*dX[n]+dX[n]*0.5;
		}
		
		Current(X, FuncParam, &tempRes);
		
		locresult[0]+=hipCreal(tempRes.x);
		locresult[1]+=hipCreal(tempRes.y);
		locresult[2]+=hipCreal(tempRes.z);
		
		locresult[3+0]+=hipCimag(tempRes.x);
		locresult[3+1]+=hipCimag(tempRes.y);
		locresult[3+2]+=hipCimag(tempRes.z);
		//printf("Re(J_res.x) %f, Im(J_res.x) %f, Re(J_res.y) %f, Im(J_res.y) %f, Re(J_res.z) %f, Im(J_res.z) %f \n",  locresult[0], locresult[0+3], locresult[1], locresult[1+3], locresult[2], locresult[2+3]);
	}
	
	for(int n=0;n<multiplicity;n++)
		{	for (int i=0; i<count; i++)
			locresult[i]*=dX[n];
			
		}
		
		//if(FuncParam[1]>2.7)
		//printf("theta %f, phi %f, Re(J_res.x) %f, Im(J_res.x) %f, Re(J_res.y) %f, Im(J_res.y) %f, Re(J_res.z) %f, Im(J_res.z) %f \n", FuncParam[1], FuncParam[2],  locresult[0], locresult[0+3], locresult[1], locresult[1+3], locresult[2], locresult[2+3]);
		//printf("%f \n", locresult[0]);
//__syncthreads();//синхронизация нитей
		//printf("%g + I %g \n", locresult[0], locresult[1] );

	for (int i=0; i< 6; i++)
		atomicAdd(&shRes[i],locresult[i]);
		
		__syncthreads();//синхронизация нитей

//	for(int thr=threadIdx.x;thr<count; thr+=blockDim.x)
//		atomicAdd(&result[thr], shRes[thr]);
//	printf("%d,  %g + I %g \n", blockIdx.x,shRes[0], shRes[1] );


	if(threadIdx.x<6)
		{
		atomicAdd(&result[threadIdx.x],shRes[threadIdx.x]);
		}
//	for (int i=0; i< 6; i++)
//		atomicAdd(&result[i],locresult[i]);

delete X;
delete ind;
delete locresult;
	__syncthreads();
	
	//if(threadIdx.x==0)
//printf(" Re(J_res.x) %f, Im(J_res.x) %f, Re(J_res.y) %f, Im(J_res.y) %f, Re(J_res.z) %f, Im(J_res.z) %f \n",  result[0], result[0+3], result[1], result[1+3], result[2], result[2+3]);
}



//extern "C" void CopyToConstantMem(int ParamNum,FLOAT *FuncConstParam)
//{
	//hipMemcpyToSymbol(HIP_SYMBOL(co_FuncParam),FuncConstParam,ParamNum*sizeof(FLOAT));

//}

void ParallelNquadIntegrator(int multiplicity, FLOAT *IntParams, int NumParam, FLOAT *FuncParam, FLOAT *J_res)
//J_res[CompCount]
{
	int Nblocks=NBLOCKS;
	int Nthreads=NTHREADS;
	
	int count=CompCount;
	FLOAT *dev_IntParams, *dev_LeftLim;
	FLOAT *dev_FuncParam;
	
	int *dev_TotalN,*cpu_TotalN;
	FLOAT *dev_dX,*cpu_dX, *cpu_LeftLim;
	
	FLOAT *dev_J;

    hipMalloc(&dev_TotalN, (multiplicity+1)*sizeof(int)); //выделяет (multiplicity+1)*sizeof(int) памяти на девайсе под dev_TotalN (возвращает указатель на участок памяти)
    hipMalloc(&dev_dX, multiplicity*sizeof(FLOAT));
    hipMalloc(&dev_LeftLim, multiplicity*sizeof(FLOAT));
    
    cpu_TotalN=new int[multiplicity+1]; //массив из общего числа отрезков N1*N2*N3, N2*N3, N3...
    cpu_dX=new FLOAT[multiplicity];
    cpu_LeftLim=new FLOAT[multiplicity]; //элементарные отрезки по каждому интегралу
    cpu_TotalN[0]=1;

	for(int i=0;i<multiplicity;i++)
		cpu_TotalN[0]*=IntParams[2+i*3]; //N1*N2*N3

	for(int i=0;i<multiplicity;i++)
	{
		cpu_dX[i]=(IntParams[1+i*3]-IntParams[0+i*3])/IntParams[2+i*3]; //(правый-левый)/количество = элементарный отрезок
		cpu_TotalN[i+1]=cpu_TotalN[i]/IntParams[2+i*3]; //N2*N3, N3...
		cpu_LeftLim[i]=IntParams[i*3]; //левый предел
		//printf("LeftLim %f \n",cpu_LeftLim[i]);
	}
	
    hipMemcpy( dev_dX, cpu_dX,  multiplicity*sizeof(FLOAT), hipMemcpyDefault); //копирует multiplicity*sizeof(FLOAT) байт из cpu_dX в dev_dX
    hipMemcpy( dev_TotalN, cpu_TotalN,  (multiplicity+1)*sizeof(int), hipMemcpyDefault);
    hipMemcpy( dev_LeftLim, cpu_LeftLim,  multiplicity*sizeof(FLOAT), hipMemcpyDefault); 

    
    hipMalloc(&dev_IntParams, multiplicity*3*sizeof(FLOAT));
    hipMalloc(&dev_FuncParam, NumParam*sizeof(cuFLOATComplex));
    
    hipMemcpy( dev_IntParams, IntParams,  multiplicity*3*sizeof(FLOAT), hipMemcpyDefault);
    hipMemcpy( dev_FuncParam, FuncParam,  NumParam*sizeof(cuFLOATComplex), hipMemcpyDefault);
    
    hipMalloc(&dev_J, count*sizeof(FLOAT));
    hipMemset(dev_J,0,count*sizeof(FLOAT)); //заполняет 1*sizeof(FLOAT) байт  в dev_Result значением 0


 //вызов ядра
 
	//printf("Kernel start");
 
	KernelParallelIntegrator<<<Nblocks, Nthreads>>>(dev_J, multiplicity, dev_IntParams, dev_FuncParam, dev_dX, dev_TotalN, dev_LeftLim);
	gpuErrchk( hipPeekAtLastError() );
	//printf("Kernel finish");
	hipDeviceSynchronize(); 

	//hipDeviceSynchronize(); //синхронизация
    hipMemcpy(J_res, dev_J,  count*sizeof(FLOAT), hipMemcpyDefault);

hipFree(dev_IntParams); //освобождение выделенной памяти
hipFree(dev_FuncParam);
hipFree(dev_J);
hipFree(dev_LeftLim);

hipFree(dev_TotalN);
delete cpu_TotalN;
hipFree(dev_dX),
delete cpu_dX;
delete cpu_LeftLim;

}

